#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
using namespace std;
__global__ void vector_add(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
		C[i] = A[i] + B[i];
		printf("Thread ID: %d (blockIdx: %d, threadIdx: %d)\n", i, blockIdx.x, threadIdx.x);
	}
}

int main() {
    int N = 1 << 12; 
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // 초기화
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2*i;
    }

    // GPU 메모리 할당
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // CPU -> GPU 복사
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	
    // GPU에서 커널 실행
    int threadsPerBlock = 256; // 한 블럭에서 실행되는 thread
    int blocksPerGrid = N/threadsPerBlock ; // 전체 블록 수
	cout << "block per grid : "<<blocksPerGrid << endl;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N); // 블록당 256개 스레드로 실행

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	}

    // GPU -> CPU 복사
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 결과
	std::cout<< "C[100] = " << h_C[100] << std::endl;

	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
